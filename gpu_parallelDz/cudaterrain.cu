#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "cudaterrain.h"



__device__ CustomPoint deviceRotate(CustomPoint p, float phi) {
    float xtemp = p.x * cos(phi) + p.y * sin(phi);
    float ytemp = p.x * -sin(phi) + p.y * cos(phi);
    return (CustomPoint){xtemp, ytemp};
}


__global__ void CalculateDepthsKernel(
    DepthPixel* depth_buffer,  // WIDTH * num_depths size
    const uint8_t* heightmap,
    const uint32_t* colormap,
    float p_x, float p_y,
    float phi,
    float height,
    int num_depths)  // Current z value
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    if (x >= WIDTH || blockIdx.y * blockDim.y + threadIdx.y >= num_depths) return;
    float z = 5.0f + blockIdx.y * 0.1f;

    float scale = -1.0f / z * 240.0f;
    float offset = -height;
    float horizon = 100;

    // Calculate points for this z
    CustomPoint pl = {-z, -z};
    CustomPoint pr = {z, -z};

    pl = deviceRotate(pl, phi);
    pr = deviceRotate(pr, phi);

    CustomPoint p1 = {p_x + pl.x, p_y + pl.y};
    CustomPoint p2 = {p_x + pr.x, p_y + pr.y};

    float dx = (p2.x - p1.x) / WIDTH;
    float dy = (p2.y - p1.y) / WIDTH;

    float cur_x = p1.x + dx * x;
    float cur_y = p1.y + dy * x;
    
    int xi = ((int)floor(cur_x)) & (MAP_SIZE - 1);
    int yi = ((int)floor(cur_y)) & (MAP_SIZE - 1);

    float heightonscreen = (heightmap[yi * MAP_SIZE + xi] + offset) * scale + horizon;
    
    int depth_idx = (blockIdx.y * blockDim.y + threadIdx.y)* WIDTH + x;
    depth_buffer[depth_idx].point_height = heightonscreen;
    depth_buffer[depth_idx].color = colormap[yi * MAP_SIZE + xi];
}


__global__ void MergeDepthsKernel(
    uint32_t* screen,
    float* hidden,
    DepthPixel* depth_buffer,
    int num_depths)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    if (x >= WIDTH) return;
    for(int y = 0; y < HEIGHT; y++) {
        screen[y * WIDTH + x] = BLUE;
    }
    hidden[x] = HEIGHT;
    
    //Process all depths for this x coordinate
    for (int d = 0; d < num_depths; d++) {
        DepthPixel pixel = depth_buffer[d * WIDTH + x];
        //if (!pixel.valid) continue;
        
        if (pixel.point_height < hidden[x]) {
            int ytop = (int)pixel.point_height;
            int ybottom = (int)hidden[x];
            
            if (ytop < 0) ytop = 0;
            if (ybottom > HEIGHT) ybottom = HEIGHT;
            
            for (int y = ytop; y < ybottom; y++) {
                screen[y * WIDTH + x] = pixel.color;
            }
            hidden[x] = pixel.point_height;
        }
    }
}

void launchRenderKernel(
    uint32_t* d_screen,
    float* d_hidden,
    const uint8_t* d_heightmap,
    const uint32_t* d_colormap,
    float p_x, float p_y,
    float phi,
    float height,
    float distance)
{
    // Calculate number of z steps
    int num_depths = (int)((distance - 5.0f) / DZ);
    
    // Allocate depth buffer
    //Can be moved to initCudaMemory
    DepthPixel* d_depth_buffer;
    cudaCheckError(hipMalloc(&d_depth_buffer, WIDTH * num_depths * sizeof(DepthPixel)));
    
    // Launch CalculateDepthsKernel
    // All depths are calculated in parallel
    dim3 threadsPerBlock(256, 2);
    dim3 numBlocks((WIDTH + threadsPerBlock.x - 1) / threadsPerBlock.x, (num_depths + threadsPerBlock.y - 1) / threadsPerBlock.y);
    CalculateDepthsKernel<<<numBlocks, threadsPerBlock>>>(
        d_depth_buffer, d_heightmap, d_colormap, p_x, p_y, phi, height, num_depths); 

    hipDeviceSynchronize();
    // Launch merge kernel
    int mergeThreads = 256;
    int mergeBlocks = (WIDTH + mergeThreads - 1) / mergeThreads;
    MergeDepthsKernel<<<mergeBlocks, mergeThreads>>>(
        d_screen, d_hidden, d_depth_buffer, num_depths);
    hipDeviceSynchronize();
    // can be freed after all screen are rendered
    hipFree(d_depth_buffer);
}



// Memory initialization
void initCudaMemory(
    uint32_t** d_screen,
    float** d_hidden,
    uint8_t** d_heightmap,
    uint32_t** d_colormap)
{
    // Allocate device memory
    cudaCheckError(hipMalloc(d_screen, WIDTH * HEIGHT * sizeof(uint32_t)));
    cudaCheckError(hipMalloc(d_hidden, WIDTH * sizeof(float)));
    cudaCheckError(hipMalloc(d_heightmap, MAP_SIZE * MAP_SIZE * sizeof(uint8_t)));
    cudaCheckError(hipMalloc(d_colormap, MAP_SIZE * MAP_SIZE * sizeof(uint32_t)));

    // Initialize hidden buffer with HEIGHT
    float init_value = HEIGHT;
    cudaCheckError(hipMemset(*d_hidden, init_value, WIDTH * sizeof(float)));
    
    // Initialize screen with sky color (optional)
    uint32_t sky_color = 0x87CEEB;
    cudaCheckError(hipMemset(*d_screen, sky_color, WIDTH * HEIGHT * sizeof(uint32_t)));
}

// Copy data to GPU
void copyDataToGPU(
    uint8_t* d_heightmap, const uint8_t heightmap[MAP_SIZE][MAP_SIZE],
    uint32_t* d_colormap, const uint32_t colormap[MAP_SIZE][MAP_SIZE])
{
    // Copy heightmap to GPU
    cudaCheckError(hipMemcpy(d_heightmap, 
                             heightmap, 
                             MAP_SIZE * MAP_SIZE * sizeof(uint8_t),
                             hipMemcpyHostToDevice));

    // Copy colormap to GPU
    cudaCheckError(hipMemcpy(d_colormap, 
                             colormap, 
                             MAP_SIZE * MAP_SIZE * sizeof(uint32_t),
                             hipMemcpyHostToDevice));
}

// Free CUDA memory
void freeCudaMemory(
    uint32_t* d_screen,
    float* d_hidden,
    uint8_t* d_heightmap,
    uint32_t* d_colormap)
{
    if (d_screen) cudaCheckError(hipFree(d_screen));
    if (d_hidden) cudaCheckError(hipFree(d_hidden));
    if (d_heightmap) cudaCheckError(hipFree(d_heightmap));
    if (d_colormap) cudaCheckError(hipFree(d_colormap));
}

