#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "cudaterrain.h"
#include <iostream>
#include <chrono>



__device__ CustomPoint deviceRotate(CustomPoint p, float phi) {
    float xtemp = p.x * cos(phi) + p.y * sin(phi);
    float ytemp = p.x * -sin(phi) + p.y * cos(phi);
    return (CustomPoint){xtemp, ytemp};
}


__global__ void CalculateDepthsKernel(
    DepthPixel* depth_buffer,  // WIDTH * num_depths size
    const uint8_t* heightmap,
    const uint32_t* colormap,
    float p_x, float p_y,
    float phi,
    float height,
    int num_depths)  // Current z value
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    if (x >= WIDTH || blockIdx.y * blockDim.y + threadIdx.y >= num_depths) return;
    float z = 5.0f + blockIdx.y * 0.1f;

    float scale = -1.0f / z * 240.0f;
    float offset = -height;
    float horizon = 100;

    // Calculate points for this z
    CustomPoint pl = {-z, -z};
    CustomPoint pr = {z, -z};

    pl = deviceRotate(pl, phi);
    pr = deviceRotate(pr, phi);

    CustomPoint p1 = {p_x + pl.x, p_y + pl.y};
    CustomPoint p2 = {p_x + pr.x, p_y + pr.y};

    float dx = (p2.x - p1.x) / WIDTH;
    float dy = (p2.y - p1.y) / WIDTH;

    float cur_x = p1.x + dx * x;
    float cur_y = p1.y + dy * x;
    
    int xi = ((int)floor(cur_x)) & (MAP_SIZE - 1);
    int yi = ((int)floor(cur_y)) & (MAP_SIZE - 1);

    float heightonscreen = (heightmap[yi * MAP_SIZE + xi] + offset) * scale + horizon;
    
    int depth_idx = (blockIdx.y * blockDim.y + threadIdx.y)* WIDTH + x;
    depth_buffer[depth_idx].point_height = heightonscreen;
    depth_buffer[depth_idx].color = colormap[yi * MAP_SIZE + xi];
}


__global__ void MergeDepthsKernel(
    uint32_t* screen,
    DepthPixel* depth_buffer,
    int num_depths)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    if (x >= WIDTH) return;
    for(int y = 0; y < HEIGHT; y++) {
        screen[y * WIDTH + x] = BLUE;
    }
    float hidden = HEIGHT;
    
    //Process all depths for this x coordinate
    for (int d = 0; d < num_depths; d++) {
        DepthPixel pixel = depth_buffer[d * WIDTH + x];
        //if (!pixel.valid) continue;
        
        if (pixel.point_height < hidden) {
            int ytop = (int)pixel.point_height;
            int ybottom = (int)hidden;
            
            if (ytop < 0) ytop = 0;
            if (ybottom > HEIGHT) ybottom = HEIGHT;
            
            for (int y = ytop; y < ybottom; y++) {
                screen[y * WIDTH + x] = pixel.color;
            }
            hidden = pixel.point_height;
        }
    }
}

void launchRenderKernel(
    uint32_t** d_screens,  // Array of screen buffers for each frame
    const uint8_t* d_heightmap,
    const uint32_t* d_colormap,
    float initial_p_x, float initial_p_y,  // Initial camera positions
    float phi,
    float height,
    float distance,
    int num_frames)  // Number of frames to render
{
    int num_depths = (int)((distance - 5.0f) / DZ);
    
    // Allocate depth buffer for all frames
    DepthPixel* d_depth_buffer;
    cudaCheckError(hipMalloc(&d_depth_buffer, num_frames * WIDTH * num_depths * sizeof(DepthPixel)));

    // Create streams
    hipStream_t streams[num_frames];
    for (int i = 0; i < num_frames; i++) {
        cudaCheckError(hipStreamCreate(&streams[i]));
    }

    // Launch CalculateDepthsKernel and MergeDepthsKernel for each frame
    dim3 threadsPerBlock(256, 2);
    dim3 numBlocks((WIDTH + threadsPerBlock.x - 1) / threadsPerBlock.x, (num_depths + threadsPerBlock.y - 1) / threadsPerBlock.y);
    // timer 
    auto start = std::chrono::high_resolution_clock::now();
    for (int frame_index = 0; frame_index < num_frames; frame_index++) {
        // Calculate new p_x and p_y for the current frame
        float p_x = initial_p_x;  // You can modify this based on your logic
        float p_y = initial_p_y - frame_index * 16;  // Example adjustment for p_y

        // Launch CalculateDepthsKernel
        CalculateDepthsKernel<<<numBlocks, threadsPerBlock, 0, streams[frame_index]>>>(
            d_depth_buffer + frame_index * WIDTH * num_depths, d_heightmap, d_colormap, p_x, p_y, phi, height, num_depths);
    }

    for (int frame_index = 0; frame_index < num_frames; frame_index++) {
        
        // Synchronize the stream to ensure CalculateDepthsKernel is complete
        cudaCheckError(hipStreamSynchronize(streams[frame_index]));

        // Launch MergeDepthsKernel
        MergeDepthsKernel<<<(WIDTH + 255) / 256, 256, 0, streams[frame_index]>>>(
            d_screens[frame_index], d_depth_buffer + frame_index * WIDTH * num_depths, num_depths);
        
        // Synchronize the stream to ensure MergeDepthsKernel is complete
        //cudaCheckError(hipStreamSynchronize(streams[frame_index]));
    }

    // timer
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed = end - start;
    std::cout << "kernel Elapsed time: " << elapsed.count() << " s\n";

    // Clean up streams
    for (int i = 0; i < num_frames; i++) {
        cudaCheckError(hipStreamSynchronize(streams[i]));
        hipStreamDestroy(streams[i]);  // Clean up the stream
    }

    // Free depth buffer
    hipFree(d_depth_buffer);
}



// Memory initialization
void initCudaMemory(
    uint32_t** d_screen,
    uint8_t** d_heightmap,
    uint32_t** d_colormap)
{
    // Allocate device memory
    cudaCheckError(hipMalloc(d_screen, WIDTH * HEIGHT * sizeof(uint32_t)));
    cudaCheckError(hipMalloc(d_heightmap, MAP_SIZE * MAP_SIZE * sizeof(uint8_t)));
    cudaCheckError(hipMalloc(d_colormap, MAP_SIZE * MAP_SIZE * sizeof(uint32_t)));

    
    // Initialize screen with sky color (optional)
    uint32_t sky_color = 0x87CEEB;
    cudaCheckError(hipMemset(*d_screen, sky_color, WIDTH * HEIGHT * sizeof(uint32_t)));
}

// Copy data to GPU
void copyDataToGPU(
    uint8_t* d_heightmap, const uint8_t heightmap[MAP_SIZE][MAP_SIZE],
    uint32_t* d_colormap, const uint32_t colormap[MAP_SIZE][MAP_SIZE])
{
    // Copy heightmap to GPU
    cudaCheckError(hipMemcpy(d_heightmap, 
                             heightmap, 
                             MAP_SIZE * MAP_SIZE * sizeof(uint8_t),
                             hipMemcpyHostToDevice));

    // Copy colormap to GPU
    cudaCheckError(hipMemcpy(d_colormap, 
                             colormap, 
                             MAP_SIZE * MAP_SIZE * sizeof(uint32_t),
                             hipMemcpyHostToDevice));
}

// Free CUDA memory
void freeCudaMemory(
    uint8_t* d_heightmap,
    uint32_t* d_colormap)
{
    if (d_heightmap) cudaCheckError(hipFree(d_heightmap));
    if (d_colormap) cudaCheckError(hipFree(d_colormap));
}

