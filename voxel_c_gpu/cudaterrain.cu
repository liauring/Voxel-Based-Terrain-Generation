#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "cudaterrain.h"

__global__ void horlineHiddenKernel(
    uint32_t* screen,
    float* hidden,
    const uint8_t* heightmap,
    const uint32_t* colormap,
    float p1_x, float p1_y,
    float dx, float dy,
    float offset, float scale, float horizon,
    int width, int height, int map_size)
{
    // Each thread handles one vertical line
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= width) return;

    // Calculate position in heightmap/colormap
    float cur_x = p1_x + dx * i;
    float cur_y = p1_y + dy * i;
    
    int xi = ((int)floor(cur_x)) & (map_size - 1);
    int yi = ((int)floor(cur_y)) & (map_size - 1);

    // Calculate height and get color
    float heightonscreen = (heightmap[yi * map_size + xi] + offset) * scale + horizon;
    uint32_t color = colormap[yi * map_size + xi];

    // Draw vertical line
    int ytop = (int)heightonscreen;
    int ybottom = (int)hidden[i];
    
    if (ytop < 0) ytop = 0;
    if (ybottom > height) ybottom = height;
    
    // Draw pixels for this vertical line
    for (int y = ytop; y < ybottom; y++) {
        screen[y * width + i] = color;
    }

    // Update hidden buffer
    if (heightonscreen < hidden[i]) {
        hidden[i] = heightonscreen;
    }
}

// Host function to launch the kernel
void cudaHorlineHidden(
    uint32_t* d_screen,
    float* d_hidden,
    const uint8_t* d_heightmap,
    const uint32_t* d_colormap,
    CustomPoint p1, CustomPoint p2,
    float offset, float scale, float horizon)
{
    float dx = (p2.x - p1.x) / WIDTH;
    float dy = (p2.y - p1.y) / WIDTH;

    // Configure kernel launch
    int threadsPerBlock = 256;
    int blocks = (WIDTH + threadsPerBlock - 1) / threadsPerBlock;

    horlineHiddenKernel<<<blocks, threadsPerBlock>>>(
        d_screen, d_hidden, d_heightmap, d_colormap,
        p1.x, p1.y, dx, dy,
        offset, scale, horizon,
        WIDTH, HEIGHT, MAP_SIZE
    );
    

}

// Memory initialization
void initCudaMemory(
    uint32_t** d_screen,
    float** d_hidden,
    uint8_t** d_heightmap,
    uint32_t** d_colormap)
{
    // Allocate device memory
    cudaCheckError(hipMalloc(d_screen, WIDTH * HEIGHT * sizeof(uint32_t)));
    cudaCheckError(hipMalloc(d_hidden, WIDTH * sizeof(float)));
    cudaCheckError(hipMalloc(d_heightmap, MAP_SIZE * MAP_SIZE * sizeof(uint8_t)));
    cudaCheckError(hipMalloc(d_colormap, MAP_SIZE * MAP_SIZE * sizeof(uint32_t)));

    // Initialize hidden buffer with HEIGHT
    float init_value = HEIGHT;
    cudaCheckError(hipMemset(*d_hidden, init_value, WIDTH * sizeof(float)));
    
    // Initialize screen with sky color (optional)
    uint32_t sky_color = 0x87CEEB;
    cudaCheckError(hipMemset(*d_screen, sky_color, WIDTH * HEIGHT * sizeof(uint32_t)));
}

// Copy data to GPU
void copyDataToGPU(
    uint8_t* d_heightmap, const uint8_t heightmap[MAP_SIZE][MAP_SIZE],
    uint32_t* d_colormap, const uint32_t colormap[MAP_SIZE][MAP_SIZE])
{
    // Copy heightmap to GPU
    cudaCheckError(hipMemcpy(d_heightmap, 
                             heightmap, 
                             MAP_SIZE * MAP_SIZE * sizeof(uint8_t),
                             hipMemcpyHostToDevice));

    // Copy colormap to GPU
    cudaCheckError(hipMemcpy(d_colormap, 
                             colormap, 
                             MAP_SIZE * MAP_SIZE * sizeof(uint32_t),
                             hipMemcpyHostToDevice));
}

// Free CUDA memory
void freeCudaMemory(
    uint32_t* d_screen,
    float* d_hidden,
    uint8_t* d_heightmap,
    uint32_t* d_colormap)
{
    if (d_screen) cudaCheckError(hipFree(d_screen));
    if (d_hidden) cudaCheckError(hipFree(d_hidden));
    if (d_heightmap) cudaCheckError(hipFree(d_heightmap));
    if (d_colormap) cudaCheckError(hipFree(d_colormap));
}

